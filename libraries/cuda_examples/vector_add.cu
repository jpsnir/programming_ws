/*
 * Vector addtion using cuda and cpu
 * @ author: jagatpreet
 * adapted from: https://github.com/puttsk/cuda-tutorial/tree/master/tutorial01
 */
#include <cstdio>
#include <cmath>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6
__global__ void cuda_add(float *out, float *a, float *b, int n){
    for (int i =0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}
void vector_add(float *out, float *a, float *b, int n){
    for(int i = 0; i < n; i ++){
       out[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *out;

    //Reserve memory
    a = (float*)malloc(sizeof(float)*N);
    b = (float*)malloc(sizeof(float)*N);
    out = (float*)malloc(sizeof(float)*N);

    // assign values
    for (int i=0; i < N ; i++){
        a[i] = 1.0f; b[i] = 3.3f;
    }
    vector_add(out, a, b, N);


    float *d_a, *d_b, *d_out; // stores device allocation pointer
    // allocate on cuda device
    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_out, sizeof(float)*N);
    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

    cuda_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);
    for (int i=0; i < N; i++){
        //printf("output = %f", out[i]);
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("out[0] = %f \n", out[0]);
    printf("Passed \n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // deallocate host memory
    free(a);
    free(b);
    free(out);


}
