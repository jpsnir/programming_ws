#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>

int main() {

    int nDevices;
    hipError_t err = hipGetDeviceCount(&nDevices);
    // expand this program with hipGetErrorString()
    // hipGetLastError() and hipPeekAtLastError()
    // and understand in relation to hipDeviceSynchronize and asynchrnoize
    // operations
    if (err != hipSuccess)
        std::cout << " GPU device count failed" << std::endl;
    for(int i = 0; i < nDevices; i++){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout<<" Device number:" << i << "\n";
        std::cout<<" Device name:" << prop.name << "\n";
        std::cout<<" Memory clock rate (KHz):" << prop.memoryClockRate << "\n";
        std::cout<<" Memory Bus width (bits) : " << prop.memoryBusWidth << "\n";
        std::cout<<" Compute capability : " << prop.major << "." << prop.minor << "\n";
        std::cout<<" Is Multi GPU board? : " << prop.isMultiGpuBoard << "\n";
        std::cout<<" Max block per SM : " << prop.maxBlocksPerMultiProcessor << "\n";
        std::cout<<" Max threads per SM : " << prop.maxThreadsPerBlock << "\n";
        std::cout<<" Peak memory bandwidth (GB/s) : " <<
            2.0*prop.clockRate*(prop.memoryBusWidth/8)/1.0e6 << "\n";
    }
}
