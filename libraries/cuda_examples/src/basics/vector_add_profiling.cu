#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

void add_on_cpu(int n, float *x, float *y){
    for(int i=0; i < n; i++){
        y[i] = x[i] + y[i];
    }
}

// cuda kernel
__global__ void add_on_gpu(int n, float *x, float *y){
    int t_index = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;
    for(int data_index=t_index; data_index < n; data_index+=stride){
        y[data_index] = x[data_index] + y[data_index];
    }
}

int main(void){
    int N = 1<<20;
    std::cout<<"N = "<<N<<std::endl;
    //allocate on heap
    float *x = new float[N];
    float *y = new float[N];
    float *x_gpu, *y_gpu;

    hipMallocManaged(&x_gpu, N*sizeof(float));
    hipMallocManaged(&y_gpu,N*sizeof(float));
    int value, dev_id=0;
    hipDeviceGetAttribute(&value, hipDeviceAttributeMultiprocessorCount, dev_id);
    std::cout << "Number of streaming processors : " << value << std::endl;
    hipDeviceGetAttribute(&value, hipDeviceAttributeMaxThreadsPerBlock, dev_id);
    std::cout << "Number of threads per block : " << value << std::endl;
    int blocksize = 256;
    int numblocks = N / blocksize;
    std::cout << " N = " << N << ", Number of blocks : " << numblocks << std::endl;
    for (int i=0; i < N; i++){
        x[i] = 2.0f;
        y[i] = 3.0f;
    }

    for (int i=0; i < N; i++){
        x_gpu[i] = 2.0f;
        y_gpu[i] = 3.0f;
    }

    // add elements
    add_on_cpu(N, x, y);
    hipDeviceGetAttribute(&value, hipDeviceAttributeMultiprocessorCount, dev_id);
    add_on_gpu<<<value*8, 512>>>(N, x_gpu, y_gpu);
    hipDeviceSynchronize();


    float maxError = 0.0f;
    for (int i=0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 5.0f));
    std::cout << "Max error (cpu) : " << maxError << std::endl;

    for (int i=0; i < N; i++)
        maxError = fmax(maxError, fabs(y_gpu[i] - 5.0f));
    std::cout << "Max error (gpu) : " << maxError << std::endl;
    // free memory
    delete [] x;
    delete [] y;

    hipFree(x_gpu);
    hipFree(y_gpu);

    return 0;
}
