
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    hipblasStatus_t status;
    hipblasHandle_t handle;

    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CuBLAS initialization failed." << std::endl;
        return 1;
    }

    float x = 2.0f;
    status = hipblasSscal(handle, 1, &x, &x, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CuBLAS function call failed." << std::endl;
        hipblasDestroy(handle); // Clean up before exiting
        return 1;
    }

    std::cout << "Scaled value: " << x << std::endl;

    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CuBLAS destruction failed." << std::endl;
        return 1;
    }

    return 0;
}
