#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdlib>

__global__ void addVector(float *x, float *y, float *z){
    int i = threadIdx.x;
    printf("block id = %d \n", blockIdx.x);
    printf("thread Idx = %d \n", i);
    z[i] = x[i] + y[i];
}
__global__ void addMatrix(float *x, float *y, float *z){
    int row = threadIdx.x;
    int col = threadIdx.y;

    printf("thread Idx = %d, %d \n", row, col);
    int index = row*16 + col;
    z[index] = x[index] + y[index];
}

int main(void){
    // pointers to the host data.
    float *x, *y, *z;

    // pointers to the device data
    float *d_x, *d_y, *d_z;

    // No. data elements to be processed
    int N = 1<<8; // 256
    std::cout << " N = " << N << std::endl;

    //assign memory in the device
    // ? not sure whether it will use global memory, DRAM etc. on GPU.
    hipMalloc((void**)&d_x, sizeof(float)*N);
    hipMalloc((void**)&d_y, sizeof(float)*N);
    hipMalloc((void**)&d_z, sizeof(float)*N);

    // new returns a pointer of type float
    // to first element
    //
    // we could have used malloc function, which returns a null pointer.
  //  x = new float(N);
  //  y = new float(N);
  //  z = new float(N);
    x = (float*)malloc(sizeof(float)*N);
    y = (float*)malloc(sizeof(float)*N);
    z = (float*)malloc(sizeof(float)*N);
    for(int i = 0; i < N; i ++){
        x[i] = 1.0f;
        y[i] = 1.0f;
    }
    // copy data to allocated variables on GPU.
    hipMemcpy(d_x, x, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float)*N, hipMemcpyHostToDevice);

    // In this example, we are using
    // memory on the GPU device for computation.
    addVector<<<1, 256>>>(d_x, d_y, d_z);

    // 2d block of threads
    dim3 threadMatBlock(16, 16);
    addMatrix<<<1, threadMatBlock>>>(d_x, d_y, d_z);

    // copy result back
    hipMemcpy(z, d_z, sizeof(float)*N, hipMemcpyDeviceToHost);
    std::cout << " Z[10] " << z[10] << std::endl;
    std::cout << " Z[100] " << z[100] << std::endl;
    std::cout << " Z[255] " << z[255] << std::endl;
    hipFree(d_x); hipFree(d_y); hipFree(d_z);
    exit(0);
}
