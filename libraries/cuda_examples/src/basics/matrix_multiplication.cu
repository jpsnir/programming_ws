#include <iostream>
#include <hip/hip_runtime.h>

#include <cstdio>


/* Some important highlights of this example:
 * Tells you some essential steps of the example to figure out the process of
 *    implementation of any cuda algorithm.
 *
 * 1. First thing to figure out is memory allocation of the data.
 *    Sometimes a single mathematical operation need to be performed on the
 *    data. This type is the best example for parallel programming.
 *
 *    Sometimes the data interaction can be parallelized such as matrix
 *    multiplication and other operations. This is also a pattern.
 *
 * 2. Next thing to decide is based on what operation is to be parallelized.
 *    Here in this example each (i,j) element of the output matrix is computed in
 *    parallel.
 *
 * 3. Once it is figured out which operation is to parallelize, we need to find
 *    out how many threads we need - this will also dictate the block size and
 *    grid size, should we use 1D, 2D or 3D grids of threads to compute
 *
 * 4. Figure out the elementary operation for each thread to be processed in the
 *    cuda kernel. This will iterate back to step 2 if a result cannot be found.
 *
 * 5. Implement and test. - can happen while you are thinking each step on
 *    paper.
 *
 */
typedef struct{
    int width;
    int height;
    float *elements;
}Matrix;



#define BLOCK_SIZE 16
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);


void Matmul(const Matrix A, const Matrix B, Matrix C){
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    size_t size = A.width*A.height*sizeof(float);
    hipMalloc(&d_A.elements, size); // this part is still confusing.
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    size = B.width*A.height*sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements,B.elements, size,hipMemcpyHostToDevice);

    //Allocate C
    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    size = C.width*C.height*sizeof(float);
    hipMalloc(&d_C.elements, size);

    //kernel
    // For a block size of 16 in each dimension
    // we will have 256 threads in a single block
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // no. of threads in each block

    // this defines the size of the block,
    // based on the size of C. The number of rows in
    // A - height(m) x width(n), B - height(n) x width(p) , C - mxp
    // Number of threads for rows or height of C should be M. Since each block
    // has threads of size BLOCK_SIZE,
    // we have grid dimension decided in row by A.height/BLOCK_SIZE
    // Similary, number of threads in col dimension are required to be p,
    // leading to grids in col dimension to be B.width/BLOCK_SIZE.
    //
    // NOTE: whether we assign this to grid dimension x or y is programmer's choice.

    dim3 dimGrid(B.width/dimBlock.x, A.height/dimBlock.y); // no. of blocks in
    // grid
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){
    // Total no. of threads for this computation will be
    float Cvalue = 0;
    // row and column of c matrix
    // Note that we assigned grid dim y to be row dimension
    // and grid dim x to be column dimension
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // compute element of c by multiplying a row j from A with col k of B.
    // for each element in row j and col k, get the product and add it
    for (int e=0; e<A.width; ++e){
        Cvalue += A.elements[row*A.width + e] * B.elements[e*B.width + col];
        printf("element %d %d %f",row,col, Cvalue);
    }
    C.elements[row*C.width + col] = Cvalue;
}

int main(){
    int rows = 16;
    int cols = 16;
    Matrix a = {
        .width = rows,
        .height = cols,
        .elements = (float*)malloc(rows*cols*sizeof(float))
    };
    Matrix b = {
        .width = rows,
        .height = cols,
        .elements = (float*)malloc(rows*cols*sizeof(float))
    };
    Matrix c = {
        .width = rows,
        .height = cols,
        .elements = (float*)malloc(rows*cols*sizeof(float))
    };

    for(int row=0; row < a.height ; row++){
        for (int col = 0; col < a.width; col++){
            a.elements[row*a.width + col] = col+1;
            b.elements[row*a.width + col] = 4-col;
            c.elements[row*a.width + col] = 0;
        }
    }
    Matmul(a, b, c);
    std::cout << " Matrix a " << std::endl;
    for(int row=0; row < a.height ; row++){
        for (int col = 0; col < a.width; col++){
            std::cout << a.elements[row*a.width + col] << ",";
        }
        std::cout << std::endl;
    }
    std::cout << " Matrix b " << std::endl;
    for(int row=0; row < a.height ; row++){
        for (int col = 0; col < a.width; col++){
            std::cout << b.elements[row*a.width + col] << ",";
        }
        std::cout << std::endl;
    }
    std::cout << " Matrix c " << std::endl;
    for(int row=0; row < a.height ; row++){
        for (int col = 0; col < a.width; col++){
            std::cout << c.elements[row*a.width + col] << ",";
        }
        std::cout << std::endl;
    }
}
