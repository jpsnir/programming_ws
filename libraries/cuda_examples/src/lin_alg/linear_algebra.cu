#include "hip/hip_runtime.h"
#include<cstdio>
#include<cstdlib>
#include<iostream>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hipsolver.h>
#include<hipsparse.h>
#include <vector>
#include<random>
#include <algorithm>
#include<eigen3/Eigen/Dense>
#include<eigen3/Eigen/Core>

using namespace std;
int main(){
    cout << " Using cuda for performing matrix and linear algebra operations" <<
        endl;

    // number of points
    const int N  = 10000;
    // line parameters
    float m = 2.0f, c = 2.0f;
    // noise parameters
    float noise_mean = 0;
    float noise_std = 0.1f;

    float min = -10, max = 10;
    // create data for linear regression
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> distribution(noise_mean, noise_std);
    std::uniform_int_distribution<> x_dist(min, max);

    // create data
    std::vector<float> x, y;
    x.reserve(N);
    y.reserve(N);
    for (int i = 0; i < N; i++){
        x[i] = x_dist(gen);
        y[i] = m*x[i] + c + distribution(gen);
    }


    // referring to the same data in heap
    // eigen vaectors can be directly initialized with
    // c style arrays or the map class can be to used.
    // no new memory is allocated. All these methods are
    // just mapping the std vector to be used with
    // eigen interface.
    // no copy only pointing to the data and get the interface.
    Eigen::Map<Eigen::VectorXf> X1(x.data(), N);

    // creating a copy - one line copy to eigen matrix
    Eigen::VectorXf v2 = Eigen::Map<Eigen::VectorXf>(x.data(), N);
    Eigen::VectorXf v3 = Eigen::Map<Eigen::VectorXf, Eigen::Unaligned>(x.data(), N);

    // copy back to vector
    std::vector<float> x2(v2.data(), v2.data() + v2.cols()*v2.rows());
    //Eigen::VectorXf X2(x.data(), 100);
    for (int i = 0; i < 10; i++){
        cout << " values: "<< i << ":"
            << " original = " << x[i]
            << ", mapped  = " << X1[i]
            << ", copied to eigen =" << v2[i]
            << ", copied to eigen - unaligned =" << v3[i]
            << ", copied back to vector =" << x2[i]
            << std::endl;
    }
}
