#include "hip/hip_runtime.h"
#include<cstdio>
#include<cstdlib>
#include<iostream>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hipsolver.h>
#include<hipsolver.h>
#include<hipsparse.h>
#include <vector>
#include<random>
#include <algorithm>
#include<eigen3/Eigen/Dense>
#include<eigen3/Eigen/Core>

using namespace std;
int main(int argc, char *argv[]){
    cout << " Using cuda for performing matrix and linear algebra operations" <<
        endl;
    if (argc != 2){
        cout << "usage : ./linear_algebra_eigen <N>" <<endl;
        exit(0);
    }

    // number of points
    static const int N  = std::stoi(std::string(*(argv)));
    // line parameters
    float m = 2.0f, c = 2.0f;
    // noise parameters
    float noise_mean = 0;
    float noise_std = 0.1f;

    float min = -10, max = 10;
    // create data for linear regression
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> distribution(noise_mean, noise_std);
    std::uniform_real_distribution<float> x_dist(min, max);

    // create data
    std::vector<float> x, y;
    x.reserve(N);
    y.reserve(N);
    for (int i = 0; i < N; i++){
        x[i] = x_dist(gen);
        y[i] = m*x[i] + c + distribution(gen);
    }
    // referring to the same data in heap
    // eigen vaectors can be directly initialized with
    // c style arrays using the map class can be to used.
    // no new memory is allocated.
    Eigen::Map<Eigen::VectorXf> X(x.data(), N);
    Eigen::Map<Eigen::VectorXf> Y(y.data(), N);
    Eigen::MatrixXf A;
    A.col(0)  << X;
    A.col(1) << Eigen::VectorXf::Ones(N);
    cout << A.block(0, 0, 10, 10) << endl;
}
