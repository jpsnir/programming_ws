#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

void add_on_cpu(int n, float *x, float *y){
    for(int i=0; i < n; i++){
        y[i] = x[i] + y[i];
    }
}

// cuda kernel
__global__ void add_on_gpu(int n, float *x, float *y){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i=index; i < n; i+=stride){
        y[i] = x[i] + y[i];
    }
}

int main(void){
    int N = 1<<20;
    std::cout<<"N = "<<N<<std::endl;
    //allocate on heap
    float *x = new float[N];
    float *y = new float[N];
    float *x_gpu, *y_gpu;
    hipMallocManaged(&x_gpu, N*sizeof(float));
    hipMallocManaged(&y_gpu,N*sizeof(float));

    int blocksize = 256;
    int numblocks = N / blocksize;
    std::cout << " N = " << N << ", Number of blocks : " << numblocks << std::endl;
    for (int i=0; i < N; i++){
        x[i] = 2.0f;
        y[i] = 3.0f;
    }

    for (int i=0; i < N; i++){
        x_gpu[i] = 2.0f;
        y_gpu[i] = 3.0f;
    }

    // add elements
    add_on_cpu(N, x, y);
    add_on_gpu<<<1, 256>>>(N, x_gpu, y_gpu);
    hipDeviceSynchronize();


    float maxError = 0.0f;
    for (int i=0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 5.0f));
    std::cout << "Max error (cpu) : " << maxError << std::endl;

    for (int i=0; i < N; i++)
        maxError = fmax(maxError, fabs(y_gpu[i] - 5.0f));
    std::cout << "Max error (gpu) : " << maxError << std::endl;
    // free memory
    delete [] x;
    delete [] y;

    hipFree(x_gpu);
    hipFree(y_gpu);

    return 0;
}
