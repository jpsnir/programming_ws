#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>


/* cuda program to copy matrices to cuda memory and add each element
 * We want to index the matrices with [][] notation as opposed to
 * linear referencing using single bracket []
 */

#define N 10

//cuda kernel for addition of matrix using linear memory
__global__ void  matrix_add_linear(float *a, float *b, float*c){

    //each element to be added by different thread.
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i < N*N )
        c[i] = a[i] + b[i];

}

/* initialize a square diagnoal matrix with value provided.
   global matrices are passed as pointers.
 */

void initialize_diagonal_matrix(float **h_aa, int diag_elem){
    int value;
    for (int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            if (i==j){
                value = diag_elem;
            }
            else{
                value = 0;
            }
            h_aa[i][j] = value;
        }
    }
}


int main(void){
    // allocate host memory for matrices
    int rows = N , cols = N;

    // allocate space of array in a linear fashion
    float *h_a, *h_b, *h_c;
    h_a = (float*)malloc(rows*cols*sizeof(float));
    h_b = (float*)malloc(rows*cols*sizeof(float));
    h_c = (float*)malloc(rows*cols*sizeof(float));

    for (int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            int value1, value2;
            if (i==j){
                value1 = 2;
                value2 = 3;
            }
            else{
                value1 = 0;
                value2 = 0;
            }

            h_a[rows*i+j] = value1;
            h_b[rows*i+j] = value2;
        }
    }

    float *d_a, *d_b, *d_c;
    //Allocate on cuda device
    // d_a is a pointer on host device and is
    // storing a pointer value on the GPU device
    hipMalloc((void**)&d_a,sizeof(float)*rows*cols);
    hipMalloc((void**)&d_b, sizeof(float)*rows*cols);
    hipMalloc((void**)&d_c, sizeof(float)*rows*cols);

    hipMemcpy(d_a, h_a, sizeof(float)*rows*cols,hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*rows*cols,hipMemcpyHostToDevice);
    int blocksize = N;
    int thread_numbers = N;
    matrix_add_linear<<<blocksize, thread_numbers>>>(d_a, d_b, d_c);
    hipMemcpy(h_c, d_c,sizeof(float)*rows*cols,hipMemcpyDeviceToHost);

    for(int i = 0; i < rows; i++ ){
        for(int j = 0; j < cols; j++){
            std::cout << h_c[rows*i+j] << ", " ;
        }
        std::cout << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    float **h_aa,**h_bb, **h_cc;
    initialize_diagonal_matrix(h_aa, 5);
    initialize_diagonal_matrix(h_bb, 10);
    float *d_ma, *d_mb, *d_mc;
    size_t pitch;
    hipMallocPitch(&d_ma, &pitch, cols*sizeof(float), rows);
    hipMemcpy2D(d_ma,pitch,h_aa,cols*sizeof(float),cols*sizeof(float),rows,hipMemcpyDeviceToHost);

    exit(0);

}
